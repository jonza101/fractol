#include "hip/hip_runtime.h"
#include "fractol.h"

__kernel void test();

// void	ft_start_cl(t_mlx *mlx)
// {
// 	mlx->kernel = malloc(sizeof(t_opencl));

// 	mlx->kernel->ret = clGetDeviceIDs(NULL, CL_device_ext_GPU, 1, &mlx->kernel->device_id, NULL);

// 	mlx->kernel->context = clCreateContext(0, 1, &mlx->kernel->device_id, NULL, NULL, &mlx->kernel->ret);

// 	mlx->kernel->commant_queue = clCreateCommandQueue(mlx->kernel->context, mlx->kernel->device_id, 0, &mlx->kernel->ret);
	
// 	//mlx->kernel->ret = clGetPlatformIDs(1, &mlx->kernel->platform_id, &mlx->kernel->ret_num_platform);
// }

void	ft_compile_cl(t_mlx *mlx)
{
	cl_program program;
	cl_kernel kernel;

	int fd;
	size_t source_size;
	const char *source_str;
	size_t source_len;
	char *temp;
	int i;

	fd = open("test_kernel.cl", O_RDONLY);
	temp = ft_strnew(1);
	source_str = ft_strnew(1);
	while (1)
	{
		while (get_next_line(fd, &temp) != 0)
		{
			source_str = ft_strjoin(source_str, temp);
			source_str = ft_strjoin(source_str, "\n");
		}
		break ;
	}
	close(fd);

	source_len = ft_strlen(source_str);

	printf("source len_log: %lu\n", source_len);
	printf("%s\n", source_str);


	mlx->kernel = malloc(sizeof(t_opencl));

	if ((mlx->kernel->ret = clGetDeviceIDs(NULL, CL_DEVICE_TYPE_GPU, 1, &mlx->kernel->device_id, NULL)) != CL_SUCCESS)
	{
		printf("Error0\n");
		exit(0);
	}

	if (!(mlx->kernel->context = clCreateContext(0, 1, &mlx->kernel->device_id, NULL, NULL, &mlx->kernel->ret)))//CL_CONTEXT_PLATFORM
	{
		printf("Error1\n");
		exit(0);
	}

	if (!(mlx->kernel->commant_queue = clCreateCommandQueue(mlx->kernel->context, mlx->kernel->device_id, 0, &mlx->kernel->ret)))
	{
		printf("Error2\n");
		exit(0);
	}

	if (!(program = clCreateProgramWithSource(mlx->kernel->context, 1, (const char**)&source_str, &source_len, &mlx->kernel->ret)))
	{
		printf("Error3\n");
		exit(0);
	}



	size_t len_log;
	size_t len_status;
	size_t len_options;
	char device_info[512];
	char device_ext[512];

	if ((mlx->kernel->ret = clBuildProgram(program, 1, &mlx->kernel->device_id, NULL, NULL, NULL)) != CL_SUCCESS)
	{
		printf("ret %d\n", mlx->kernel->ret);
		//printf("Kernel Error!\n");

		clGetDeviceInfo(mlx->kernel->device_id, CL_DEVICE_NAME, 512, device_info, NULL);
		clGetDeviceInfo(mlx->kernel->device_id, CL_DEVICE_EXTENSIONS, 512, device_ext, NULL);

		printf("\nDEVICE INFO: %s\n", device_info);
		printf("\nDEVICE EXT: %s\n", device_ext);

		clGetProgramBuildInfo(program, mlx->kernel->device_id, CL_PROGRAM_BUILD_LOG, 0, NULL, &len_log);
		clGetProgramBuildInfo(program, mlx->kernel->device_id, CL_PROGRAM_BUILD_STATUS, 0, NULL, &len_status);
		clGetProgramBuildInfo(program, mlx->kernel->device_id, CL_PROGRAM_BUILD_OPTIONS, 0, NULL, &len_options);
		
		char *log = (char*)malloc(len_log);
		char *status = (char*)malloc(len_status);
		char *options = (char*)malloc(len_options);

		clGetProgramBuildInfo(program, mlx->kernel->device_id, CL_PROGRAM_BUILD_LOG, len_log, log, NULL);
		clGetProgramBuildInfo(program, mlx->kernel->device_id, CL_PROGRAM_BUILD_STATUS, len_status, status, NULL);
		clGetProgramBuildInfo(program, mlx->kernel->device_id, CL_PROGRAM_BUILD_STATUS, len_options, options, NULL);

		printf("\nLOG: %s\n", log);
		printf("\nSTATUS: %s\n", status);
		printf("\nOPTIONS: %s\n", options);
		exit(0);
	}

	kernel = clCreateKernel(program, "test", &mlx->kernel->ret);



	// cl_mem mem_obj = NULL;
	// //int mem_len = 1;
	// t_mlx *mem = (t_mlx*)malloc(sizeof(t_mlx));

	// mem_obj = clCreateBuffer(mlx->kernel->context, CL_MEM_READ_WRITE,  sizeof(t_mlx), NULL, &mlx->kernel->ret);

	// mlx->kernel->ret = clEnqueueWriteBuffer(mlx->kernel->commant_queue, NULL, CL_TRUE, 0, sizeof(t_mlx), mem, 0, NULL, NULL);

	// //mlx->kernel->ret = clSetKernelArg(kernel, 0, sizeof(cl_mem), NULL);



	// mlx->kernel->ret = clEnqueueNDRangeKernel(mlx->kernel->commant_queue, kernel, 1, NULL, NULL, NULL, 0, NULL, NULL);

	// mlx->kernel->ret = clEnqueueReadBuffer(mlx->kernel->commant_queue, mem_obj, CL_TRUE, 0, sizeof(t_mlx), mem, 0, NULL, NULL);

	printf("aaa\n");
}

// CL_INVALID_PROGRAM if program is not a valid program object.
// CL_INVALID_VALUE if device_list is NULL and num_devices is greater than zero, or if device_list is not NULL and num_devices is zero.
// CL_INVALID_VALUE if pfn_notify is NULL but user_data is not NULL.
// CL_INVALID_DEVICE if OpenCL devices listed in device_list are not in the list of devices associated with program.
// CL_INVALID_BINARY if program is created with clCreateWithProgramWithBinary and devices listed in device_list do not have a valid program binary loaded.
// CL_INVALID_BUILD_OPTIONS if the build options specified by options are invalid.
// CL_INVALID_OPERATION if the build of a program executable for any of the devices listed in device_list by a previous call to clBuildProgram for program has not completed.
// CL_COMPILER_NOT_AVAILABLE if program is created with clCreateProgramWithSource and a compiler is not available i.e. CL_DEVICE_COMPILER_AVAILABLE specified in the table of OpenCL Device Queries for clGetDeviceInfo is set to CL_FALSE.
// CL_BUILD_PROGRAM_FAILURE if there is a failure to build the program executable. This error will be returned if clBuildProgram does not return until the build has completed.
// CL_INVALID_OPERATION if there are kernel objects attached to program.
// CL_OUT_OF_RESOURCES if there is a failure to allocate resources required by the OpenCL implementation on the device.
// CL_OUT_OF_HOST_MEMORY if there is a failure to allocate resources required by the OpenCL implementation on the host.